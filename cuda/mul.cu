
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstring>
#include <hip/hip_runtime_api.h>
using namespace std;

long long squaresum(long long n) {return n*(n+1)*(2*n+1)/6;}

__global__ void cudamul_kernel(long long *A, long long *B, long long *C, int N) {
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	if(id >= N * N) return;
	int i = id / N, j = id % N, k = 0;
	for(k = 0; k < N; k++) {
		C[id] += A[i * N + k] * B[k * N + j];
	}
}

int main(int argc, char **argv){
	long long *A, *B, *C;
	int N; 
	// N should lower than 2048,
	// otherwise squaresum result will be overflow.
	if(argc == 1) N = 2047;
	else N = atoi(argv[1]);
	// init
	A = new long long [N*N]();
	B = new long long [N*N]();
	C = new long long [N*N]();
	for(int i = 0; i < N; i++) {
		for(int j = 0; j < N; j++) {
			A[i * N + j] = i * N + j + 1;
			B[i * N + j] = j * N + i + 1;
		}
	}
	hipSetDevice(0);
	long long *cuda_A, *cuda_B, *cuda_C;
	int nBypes = N * N * sizeof(long long);
	hipMalloc(&cuda_A, nBypes);
	hipMalloc(&cuda_B, nBypes);
	hipMalloc(&cuda_C, nBypes);
	
	cout << "begin" << endl;
	hipMemcpy(cuda_A, A, nBypes, hipMemcpyHostToDevice);
	hipMemcpy(cuda_B, B, nBypes, hipMemcpyHostToDevice);
	hipMemcpy(cuda_C, C, nBypes, hipMemcpyHostToDevice);
	
	int threadperblock = 256;
	int blockpergrid = (N * N + threadperblock - 1) / threadperblock;
	cudamul_kernel<<<blockpergrid, threadperblock>>>(cuda_A, cuda_B, cuda_C, N);
	
	hipMemcpy(C, cuda_C, nBypes, hipMemcpyDeviceToHost);
	// cudaDeviceSynchronize();
	hipFree(cuda_A);
	hipFree(cuda_B);
	hipFree(cuda_C);
	
	cout << "end" << endl;
	cout << C[N * N - 1] << endl;
	cout << squaresum(N*N) - squaresum(N*N-N) << endl;
	return 0;
}
